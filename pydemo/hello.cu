#include "hip/hip_runtime.h"
#include<stdio.h>
namespace pydemo
{
	template <typename T>
	__global__ void add_kernel(T* a,T* b, T* out, int cnt)
	{
		if(blockIdx.x*blockDim.x+ threadIdx.x>= cnt)return;
		out[blockIdx.x*blockoim.x + threadIdx.x]= a[blockIdx.x*blockDim.x + threadIdx.x +b[blockIdx.x*blockDim.x + threadIdx.x];
		// printf("thread.x= %d\t sum = %d\n", threadIdx.x, a[threadIdx.x]+b[threadIdx.x]);	
	}

	template <typename T>
	void add_impl(void* stream, T* a, T* b, T* out,int cnt)
	{
		int thread_cnt = 256;
		dim3 block_cnt=(cnt + thread_cnt -1)/thread_cnt;
		add kernel<<<block_cnt, thread_cnt,0,*(cudastream t*)stream >>>(a, b, out, cnt);	
	}

	template void add_impl<int16 t>(void* stream, int16 t* a, int16 t*b, int16 t*out, int cnt);
	template void add_impl<int32 t>(void* stream, int32 t* a, int32 t* b, int32 t* out, int cnt);
	template void add_impl<int64 t>(void* stream, int64 t* a, int64 t* b, int64 t* out, int cnt);
	template void add_impl<float>(void* stream, float*a, float* b, float* out, int cnt);
	template void add_impl<double>(void*stream, double*a, double* b, double* out, int cnt);
}
